#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include "cuda_wrapper.h"

__device__ int mandel(double x, double y, int max_count){
  double zr = 0;
  double zrtemp;
  double zi = 0;
  int count = 0;
  while(count < max_count && zr*zr + zi*zi < 4) {
    zrtemp = zr;
    zr = zr*zr - zi*zi + x;
    zi = 2*zrtemp*zi + y;
    ++count;
  }
  return count;
}

__global__ void mandel_kernel(int *iter_counts, int max_count, int width, int height, double xmin, double ymin, double xmax, double ymax)
{
  const int N = width*height;

  float dwidth = static_cast<double>(width);
  float dheight = static_cast<double>(height);
  // printf("blockIdx.x: %d    blockDim.x: %d    threadIdx.x: %d    gridDim.x: %d\n", blockIdx.x, blockDim.x, threadIdx.x, gridDim.x);

  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < N; idx += blockDim.x * gridDim.x) 
  { 
    int xind = idx % width;
    int yind = idx / width;

    double x = xmin + xind/(dwidth-1) * (xmax - xmin);
    double y = ymin + yind/(dheight-1) * (ymax - ymin); 

    iter_counts[idx] = mandel(x, y, 255);
    //printf("Index: %d, xind: %d, yind: %d, x: %.3f, y: %.3f, counts: %d\n", idx, xind, yind, x, y, iter_counts[idx]);
  }
}

MandelKernelWrapper::MandelKernelWrapper(int width, int height, int max_count) : width_(width), height_(height), max_count_(max_count) {
  const int N = width*height;
  bytes_ = N*sizeof(int);

  // Allocate memory for arrays on host.
  iter_counts_ = (int*)malloc(bytes_);

  // Initialize memory on host.
  memset(iter_counts_, 0, bytes_);

  // Allocate memory for arrays on device.
  hipMalloc(&d_iter_counts_, bytes_);

  // Block config.
  blk_in_grid_ = ceil( float(N) / thr_per_blk_ );
}

MandelKernelWrapper::~MandelKernelWrapper() {
  // Free CPU memory.
  free(iter_counts_);

  // Free GPU memory
  hipFree(d_iter_counts_);
}

int* MandelKernelWrapper::call_kernel(double xmin, double ymin, double xmax, double ymax) {
  // Reset iter counts.
  memset(iter_counts_, 0, bytes_);

  // Copy input data from host to device. 
  hipMemcpy(d_iter_counts_, iter_counts_, bytes_, hipMemcpyHostToDevice);

  // Call mandel kernel on device. 
  mandel_kernel<<< blk_in_grid_, thr_per_blk_ >>>(d_iter_counts_, max_count_, width_, height_, xmin, ymin, xmax, ymax);

  // Copy output data from device back to host.
  hipMemcpy(iter_counts_, d_iter_counts_, bytes_, hipMemcpyDeviceToHost);
  /*
  printf("\n---------------------------\n");
  printf("__SUCCESS__\n");
  printf("---------------------------\n");
  printf("N                 = %d\n", width_*height_);
  printf("Threads Per Block = %d\n", thr_per_blk_);
  printf("Blocks In Grid    = %d\n", blk_in_grid_);
  printf("---------------------------\n\n");
  */
  return iter_counts_;
}